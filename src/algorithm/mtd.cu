#include "hip/hip_runtime.h"
#include "mtd.h"
#include "tools.h"
#include "dataProcess.h"

// ---------------------- 修改FFTShift Kernel ----------------------
__global__ void fftShiftKernel2d(hipFloatComplex* data,  int q, int k) {
    const int row_idx = blockIdx.x * blockDim.x + threadIdx.x;  // 每行一个block
    const int halfQ = q >> 1;  // 用位移替代除法

 

    if (row_idx < k) {
        // 只交换半行数据，避免重复操作
        for (int swap_pos = 0; swap_pos < halfQ; ++swap_pos) {
            const int front_idx = row_idx * q + swap_pos;
            const int back_idx = front_idx + halfQ;

            // 执行交换操作
            hipFloatComplex tmp = data[front_idx];
            data[front_idx] = data[back_idx];
            data[back_idx] = tmp;

           

            // // 计算模值（使用快速近似计算）
            // abs_result[front_idx] = sqrtf(data[front_idx].x * data[front_idx].x + data[front_idx].y * data[front_idx].y);
            // abs_result[back_idx] = sqrtf(data[back_idx].x * data[back_idx].x + data[back_idx].y * data[back_idx].y);
        }
    }
}

// ---------------------- FFTHandler 修改 ----------------------
FFTHandler::FFTHandler(int Q, int SampleNumber) : Q(Q), SampleNumber(SampleNumber) {
    // 分配设备内存
    // 嵌入式设备：hipHostAlloc(页锁定内存)
    hipMalloc(&d_data, sizeof(hipFloatComplex) * Q * SampleNumber * 2);
    d_data_in_float = d_data;
    d_data_out_float = d_data_in_float + Q * SampleNumber;

    // 创建FFT计划
    hipfftPlan1d(&plan, Q, HIPFFT_C2C, SampleNumber);
}

FFTHandler::~FFTHandler() {
    // 释放设备内存
    hipFree(d_data);

    // 释放FFT计划
    hipfftDestroy(plan);
}
// 执行函数修改（核心修改部分）
hipFloatComplex* FFTHandler::execute(__half* d_data_in, hipStream_t stream) {
    // 转换半精度到单精度（使用CUDA内核）
    dim3 block(BLOCK_SIZE);
    dim3 grid((SampleNumber + BLOCK_SIZE - 1) / BLOCK_SIZE);
    convertHalfToFloat_Optimized<<<grid, block, 0, stream>>>(d_data_in, d_data_in_float, Q * SampleNumber);
    // convertHalfToFloat<<<grid, block, 0, stream>>>(d_data_in, d_data_in_float, Q * SampleNumber);

    hipfftSetStream(plan, stream);
    // 执行FFT
    hipfftExecC2C(plan, d_data_in_float, d_data_out_float, HIPFFT_FORWARD);

    return d_data_out_float;
}

hipFloatComplex* FFTHandler::execute(hipFloatComplex* d_data_in_float, hipStream_t stream) {
    hipfftSetStream(plan, stream);
    hipfftExecC2C(plan, d_data_in_float, d_data_out_float, HIPFFT_FORWARD);  // 直接计算
    return d_data_out_float;
}


// ---------------------- FFTShift2DHandler 修改 ----------------------
FFTShift2DHandler::FFTShift2DHandler(int Q, int SampleNumber) : Q(Q), SampleNumber(SampleNumber) {
    // 分配设备内存（使用默认设备）
    hipMalloc(&d_data, sizeof(hipFloatComplex) * Q * SampleNumber);
    hipMalloc(&d_abs_result, sizeof(hipFloatComplex) * Q * SampleNumber);
}

FFTShift2DHandler::~FFTShift2DHandler() {
    // 释放设备内存
    hipFree(d_data);
    hipFree(d_abs_result);
}

hipFloatComplex* FFTShift2DHandler::execute(hipFloatComplex* d_data_in, hipStream_t stream) {
    // 如果输入数据与内部缓冲区不同，拷贝数据
    if (d_data_in != d_data) {
        hipMemcpyAsync(d_data, d_data_in, sizeof(hipFloatComplex) * Q * SampleNumber, hipMemcpyDeviceToDevice, stream);
    }

    // Set thread and block configuration for CUDA kernel
    dim3 block(256);
    dim3 grid((SampleNumber + 256 - 1) / 256);

    // Call the CUDA kernel to perform FFT shift
    fftShiftKernel2d<<<grid, block, 0, stream>>>(d_data,  Q, SampleNumber);

    return d_data;
}


void MTD_CUDA_SIM_2D_C_Style(__half* d_MFout,  hipFloatComplex* h_MTDabsout, int rows, int cols) {
    // 初始化处理器
    FFTHandler fftHandler(rows, cols);
    FFTShift2DHandler fftShift2Dhandler(rows, cols);

    // 执行流水线（全部同步到同一个流）

    hipFloatComplex* fft_result = fftHandler.execute(d_MFout);
    hipFloatComplex* d_abs_res = fftShift2Dhandler.execute(fft_result);

    // 同步并拷贝结果
    hipError_t err = hipMemcpyAsync(h_MTDabsout, d_abs_res, sizeof(hipFloatComplex) * rows * cols, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cout << "寄了" << std::endl;
    }

}

void MTD_CUDA_SIM_2D_C_Style(hipFloatComplex* d_SVDout,  hipFloatComplex* h_MTDabsout, int rows, int cols) {
    // 初始化处理器
    FFTHandler fftHandler(rows, cols);

    FFTShift2DHandler fftShift2Dhandler(rows, cols);



    // 执行流水线（全部同步到同一个流）
    hipFloatComplex* fft_result = fftHandler.execute(d_SVDout);
    hipFloatComplex* d_abs_res = fftShift2Dhandler.execute(fft_result);



    // 同步并拷贝结果
    hipError_t err = hipMemcpyAsync(h_MTDabsout, d_abs_res, sizeof(hipFloatComplex) * rows * cols, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cout << "寄了" << std::endl;
    }
}


