#include "hip/hip_runtime.h"
#include "dataProcess.h"


void ReadMatlabBin(const std::string& filename, hipFloatComplex* data, int rows, int cols)
{
    std::ifstream in(filename, std::ios::binary);
    if (!in) {
        std::cerr << "Error: Cannot open file " << filename << std::endl;
        exit(1);
    }

    // 计算预期文件大小（复数数量 x 2个float）
    const size_t expected_size = 2 * rows * cols * sizeof(float);
    in.seekg(0, std::ios::end);
    size_t actual_size = in.tellg();
    in.seekg(0, std::ios::beg);

    if (actual_size != expected_size) {
        std::cerr << "Error: File size mismatch. Expected " << expected_size
        << " bytes, got " << actual_size << " bytes." << std::endl;
        exit(1);
    }

    // 读取交替存储的实部虚部数据
    std::vector<float> interleaved_data(2 * rows * cols);
    in.read(reinterpret_cast<char*>(interleaved_data.data()), actual_size);

    // 重建复数数组（直接填充用户提供的指针）
    for (int i = 0; i < rows * cols; ++i) {
        data[i] = make_hipFloatComplex(
            interleaved_data[2*i],      // 实部
            interleaved_data[2*i + 1]   // 虚部
        );
    }

    // 验证输出
    std::cout << "First 3 elements (MATLAB compatible):\n";
    for (int i = 0; i < 3; ++i) {
      printf("[%d] (%.6f, %.6f)\n", i, data[i].x, data[i].y);
    }
}


void WriteMatlabBin(const std::string& filename,  const hipFloatComplex* data,  int rows, int cols) 
{
    // 新增指针有效性验证
    if (data == nullptr) {
        std::cerr << "Error: Data pointer is null!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // 通过维度计算数据量
    const size_t num_elements = rows * cols;

    // 维度合理性验证（替换原vector大小检查）
    if (rows <= 0 || cols <= 0) {
        std::cerr << "Error: Invalid dimensions. rows: " << rows
                 << ", cols: " << cols << std::endl;
        exit(EXIT_FAILURE);
    }

    // 打开文件（保持原有逻辑）
    std::ofstream out(filename, std::ios::binary | std::ios::trunc);
    if (!out.is_open()) {
        std::cerr << "Error: Cannot create output file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    // 构建交错数据（直接操作指针）
    std::vector<float> interleaved(2 * num_elements);
    for (size_t i = 0; i < num_elements; ++i) {
        interleaved[2*i] = data[i].x;    // 实部
        interleaved[2*i + 1] = data[i].y; // 虚部
    }

    // 写入文件（保持原有逻辑）
    out.write(reinterpret_cast<const char*>(interleaved.data()),
             interleaved.size() * sizeof(float));

    // 检查写入状态（保持原有逻辑）
    if (!out.good()) {
        std::cerr << "Error: Failed during file writing!" << std::endl;
        exit(EXIT_FAILURE);
    }
    out.close();
}



/**
 * @brief 将hipFloatComplex数组转换为Eigen Tensor
 * @param input 输入数组
 * @param output 输出Eigen Tensor
 * @param rows 行数
 * @param cols 列数
 * @param depth 深度
 */
void convert_to_eigen_tensor(const hipFloatComplex* input, Eigen::Tensor<thrust::complex<float>, 3, Eigen::ColMajor>& output, int rows, int cols, int depth) {
    output = Eigen::Tensor<thrust::complex<float>, 3, Eigen::ColMajor>(rows, cols, depth);
    for (int d = 0; d < depth; ++d) {
        for (int r = 0; r < rows; ++r) {
            for (int c = 0; c < cols; ++c) {
                int index = d * rows * cols + r * cols + c;
                thrust::complex<float> value(input[index].x, input[index].y);
                output(r, c, d) = value;
            }
        }
    }
}

// 高效转换内核（向量化，零额外内存分配）
__global__ void convertHalfToFloat_Optimized(__half* in, hipFloatComplex* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        // 一次加载 2 个 __half（实部和虚部）
        __half2 val = *reinterpret_cast<__half2*>(&in[2 * idx]);
        // 转换为 float2
        float2 f_val = __half22float2(val);
        // 写入 hipFloatComplex
        out[idx] = make_hipFloatComplex(f_val.x, f_val.y);
    }
}