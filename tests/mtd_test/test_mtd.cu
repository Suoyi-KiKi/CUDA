#include "common_headers.h"
#include "dataProcess.h"
#include "mtd.h"
#include "tools.h"

int main(void){

    int rows = 100;
    int cols = 100;
    int num_elements = rows * cols;

    const std::string InputPath = "/home/kiki/MYCUDA/Bin/ramdon.bin";
    const std::string outputPath = "/home/kiki/MYCUDA/Bin/MTDOUT.bin";

    hipFloatComplex* h_in =  (hipFloatComplex*) malloc(num_elements * sizeof(hipFloatComplex));
    hipFloatComplex* h_out = (hipFloatComplex*) malloc(num_elements * sizeof(hipFloatComplex));
    hipFloatComplex* d_in;
    hipFloatComplex* d_out;
    hipMalloc((void**)&d_in, num_elements * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_out, num_elements * sizeof(hipFloatComplex));

    ReadMatlabBin(InputPath,h_in,rows,cols);

    hipMemcpy(d_in, h_in, num_elements * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    PROFILE_START(MTD);
    MTD_CUDA_SIM_2D_C_Style(d_in,d_out,rows,cols);
    PROFILE_END(MTD, "MTD");


    hipMemcpy(h_out, d_out, num_elements * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    WriteMatlabBin(outputPath,h_out,rows,cols);

    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}