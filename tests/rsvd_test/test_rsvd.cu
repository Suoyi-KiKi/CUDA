#include "common_headers.h"
#include "dataProcess.h"
#include "rsvd.h"
#include "tools.h"

int main(void){

    int rows = 500;
    int cols = 3*800;
    int num_elements = rows * cols;

    const std::string InputPath = "/home/kiki/MYCUDA/Bin/INPUT.bin";
    const std::string outputPath = "/home/kiki/MYCUDA/Bin/RSVDOUT.bin";

    hipFloatComplex* h_in =  (hipFloatComplex*) malloc(num_elements * sizeof(hipFloatComplex));
    hipFloatComplex* h_out = (hipFloatComplex*) malloc(num_elements * sizeof(hipFloatComplex));
    hipFloatComplex* d_in;
    hipFloatComplex* d_out;
    hipMalloc((void**)&d_in, num_elements * sizeof(hipFloatComplex));
    hipMalloc((void**)&d_out, num_elements * sizeof(hipFloatComplex));

    ReadMatlabBin(InputPath,h_in,rows,cols);

    hipMemcpy(d_in, h_in, num_elements * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    PROFILE_START(rsvd);
    de_clutter_rsvd(d_in, d_out, rows, cols, 1, 10, 2, 2);
    PROFILE_END(rsvd, "RSVD");


    hipMemcpy(h_out, d_out, num_elements * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    WriteMatlabBin(outputPath,h_out,rows,cols);

    free(h_in);
    free(h_out);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}